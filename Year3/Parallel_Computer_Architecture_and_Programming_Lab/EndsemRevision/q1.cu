#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void transformMatrix(int *A, int *B, int *rowSums, int *colSums, int m, int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n)
    {
        int val = A[row * n + col];
        if (val % 2 == 0)
            B[row * n + col] = rowSums[row];
        else
            B[row * n + col] = colSums[col];
    }
}

int main()
{
    int m, n;
    printf("Enter number of rows (M): ");
    scanf("%d", &m);
    printf("Enter number of columns (N): ");
    scanf("%d", &n);

    int A[100][100], B[100][100];
    int rowSums[100] = {0}, colSums[100] = {0};

    printf("Enter elements of %dx%d matrix A:\n", m, n);
    for (int i = 0; i < m; ++i)
        for (int j = 0; j < n; ++j)
        {
            scanf("%d", &A[i][j]);
            rowSums[i] += A[i][j];
            colSums[j] += A[i][j];
        }

    // Device pointers
    int *d_A, *d_B, *d_rowSums, *d_colSums;

    hipMalloc((void **)&d_A, m * n * sizeof(int));
    hipMalloc((void **)&d_B, m * n * sizeof(int));
    hipMalloc((void **)&d_rowSums, m * sizeof(int));
    hipMalloc((void **)&d_colSums, n * sizeof(int));

    hipMemcpy(d_A, A, m * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_rowSums, rowSums, m * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_colSums, colSums, n * sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((n + 15) / 16, (m + 15) / 16);

    transformMatrix<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_rowSums, d_colSums, m, n);

    hipMemcpy(B, d_B, m * n * sizeof(int), hipMemcpyDeviceToHost);

    printf("\nMatrix B (Result):\n");
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < n; ++j)
            printf("%d ", B[i][j]);
        printf("\n");
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_rowSums);
    hipFree(d_colSums);

    return 0;
}